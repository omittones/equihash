#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp

#define XINTREE
#define UNROLL
#define htole32(x) (x)
#define HAVE_DECL_HTOLE32 1

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include <functional>
#include <vector>

#include "equi.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "blake2b.cu"

typedef uint16_t u16;
typedef uint64_t u64;

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

#ifndef RESTBITS
#define RESTBITS	4
#endif

// 2_log of number of buckets
#define BUCKBITS (DIGITBITS-RESTBITS)

#ifndef SAVEMEM
#if RESTBITS == 4
// can't save memory in such small buckets
#define SAVEMEM 1
#elif RESTBITS >= 8
// take advantage of law of large numbers (sum of 2^8 random numbers)
// this reduces (200,9) memory to under 144MB, with negligible discarding
#define SAVEMEM 9/14
#endif
#endif

// number of buckets
static const u32 NBUCKETS = 1<<BUCKBITS;
// 2_log of number of slots per bucket
static const u32 SLOTBITS = RESTBITS+1+1;
static const u32 SLOTRANGE = 1<<SLOTBITS;
// number of slots per bucket
static const u32 NSLOTS = SLOTRANGE * SAVEMEM;
// SLOTBITS mask
static const u32 SLOTMASK = SLOTRANGE-1;
// number of possible values of xhash (rest of n) bits
static const u32 NRESTS = 1<<RESTBITS;
// RESTBITS mask
static const u32 RESTMASK = NRESTS-1;
// number of blocks of hashes extracted from single 512 bit blake2b output
static const u32 NBLOCKS = (NHASHES+HASHESPERBLAKE-1)/HASHESPERBLAKE;
// nothing larger found in 100000 runs
static const u32 MAXSOLS = 8;

// tree node identifying its children as two different slots in
// a bucket on previous layer with the same rest bits (x-tra hash)
struct tree {
  u32 bid_s0_s1_x; // manual bitfields

  __device__ tree(const u32 idx, const u32 xh) {
    bid_s0_s1_x = idx << RESTBITS | xh;
  }
  __device__ tree(const u32 idx) {
    bid_s0_s1_x = idx;
  }
#ifdef XINTREE
  __device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
  bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
  __device__ tree(const u32 bid, const u32 s0, const u32 s1) {
  bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
  }
  __device__ u32 getindex() const {
#ifdef XINTREE
    return bid_s0_s1_x >> RESTBITS;
#else
    return bid_s0_s1_x;
#endif
  }
  __device__ u32 bucketid() const {
#ifdef XINTREE
    return bid_s0_s1_x >> (2 * SLOTBITS + RESTBITS);
#else
    return bid_s0_s1_x >> (2 * SLOTBITS);
#endif
  }
  __device__ u32 slotid0() const {
#ifdef XINTREE
    return (bid_s0_s1_x >> SLOTBITS+RESTBITS) & SLOTMASK;
#else
    return (bid_s0_s1_x >> SLOTBITS) & SLOTMASK;
#endif
  }
  __device__ u32 slotid1() const {
#ifdef XINTREE
    return (bid_s0_s1_x >> RESTBITS) & SLOTMASK;
#else
    return bid_s0_s1_x & SLOTMASK;
#endif
  }
  __device__ u32 xhash() const {
    return bid_s0_s1_x & RESTMASK;
  }
};

union hashunit {
  u32 word;
  uchar bytes[sizeof(u32)];
};

#define WORDS(bits)     ((bits + 31) / 32)
#define HASHWORDS0 WORDS(WN - DIGITBITS + RESTBITS)
#define HASHWORDS1 WORDS(WN - 2*DIGITBITS + RESTBITS)

struct slot0 {
  tree attr;
  hashunit hash[HASHWORDS0];
};

struct slot1 {
  tree attr;
  hashunit hash[HASHWORDS1];
};

// a bucket is NSLOTS treenodes
typedef slot0 bucket0[NSLOTS];
typedef slot1 bucket1[NSLOTS];
// the N-bit hash consists of K+1 n-bit "digits"
// each of which corresponds to a layer of NBUCKETS buckets
typedef bucket0 digit0[NBUCKETS];
typedef bucket1 digit1[NBUCKETS];

// size (in bytes) of hash in round 0 <= r < WK
u32 hhashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
#else
  const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
#endif
  return (hashbits + 7) / 8;
}
// size (in bytes) of hash in round 0 <= r < WK
__device__ u32 hashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
#else
  const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
#endif
  return (hashbits + 7) / 8;
}

u32 hhashwords(u32 bytes) {
  return (bytes + 3) / 4;
}

__device__ u32 hashwords(u32 bytes) {
  return (bytes + 3) / 4;
}

// manages hash and tree data
struct htalloc {
  bucket0 *trees0[(WK+1)/2];
  bucket1 *trees1[WK/2];
};

typedef u32 bsizes[NBUCKETS];

struct equi {
  blake2b_state blake_ctx;
  htalloc hta;
  bsizes *nslots;
  proof *sols;
  u32 nsols;
  u32 nthreads;
  equi(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(const char *header, const u32 hlen, const char *nonce, const u32 nlen) {
    setheader(&blake_ctx, header, hlen, nonce, nlen);
    checkCudaErrors(hipMemset(nslots, 0, NBUCKETS * sizeof(u32)));
    nsols = 0;
  }
  __device__ u32 getnslots0(const u32 bid) {
    u32 &nslot = nslots[0][bid];
    const u32 n = min(nslot, NSLOTS);
    nslot = 0;
    return n;
  }
  __device__ u32 getnslots1(const u32 bid) {
    u32 &nslot = nslots[1][bid];
    const u32 n = min(nslot, NSLOTS);
    nslot = 0;
    return n;
  }
  __device__ void orderindices(u32 *indices, u32 size) {
    if (indices[0] > indices[size]) {
      for (u32 i=0; i < size; i++) {
        const u32 tmp = indices[i];
        indices[i] = indices[size+i];
        indices[size+i] = tmp;
      }
    }
  }
  __device__ void listindices1(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[0][t.bucketid()];
    const u32 size = 1 << 0;
    indices[0]    = buck[t.slotid0()].attr.getindex();
    indices[size] = buck[t.slotid1()].attr.getindex();
    orderindices(indices, size);
  }
  __device__ void listindices2(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[0][t.bucketid()];
    const u32 size = 1 << 1;
    listindices1(buck[t.slotid0()].attr, indices);
    listindices1(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices3(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[1][t.bucketid()];
    const u32 size = 1 << 2;
    listindices2(buck[t.slotid0()].attr, indices);
    listindices2(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices4(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[1][t.bucketid()];
    const u32 size = 1 << 3;
    listindices3(buck[t.slotid0()].attr, indices);
    listindices3(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices5(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[2][t.bucketid()];
    const u32 size = 1 << 4;
    listindices4(buck[t.slotid0()].attr, indices);
    listindices4(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }

#if WK == 9
  __device__ void listindices6(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[2][t.bucketid()];
    const u32 size = 1 << 5;
    listindices5(buck[t.slotid0()].attr, indices);
    listindices5(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices7(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[3][t.bucketid()];
    const u32 size = 1 << 6;
    listindices6(buck[t.slotid0()].attr, indices);
    listindices6(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices8(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[3][t.bucketid()];
    const u32 size = 1 << 7;
    listindices7(buck[t.slotid0()].attr, indices);
    listindices7(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices9(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[4][t.bucketid()];
    const u32 size = 1 << 8;
    listindices8(buck[t.slotid0()].attr, indices);
    listindices8(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
#endif
  __device__ void candidate(const tree t) {
    proof prf;
#if WK==9
    listindices9(t, prf);
#elif WK==5
    listindices5(t, prf);
#else
#error not implemented
#endif
    if (probdupe(prf))
      return;
    u32 soli = atomicAdd(&nsols, 1);
    if (soli < MAXSOLS)
#if WK==9
      listindices9(t, sols[soli]);
#elif WK==5
      listindices5(t, sols[soli]);
#else
#error not implemented
#endif
  }
  void showbsizes(u32 r) {
#if defined(HIST) || defined(SPARK) || defined(LOGSPARK)
    u32 ns[NBUCKETS];
    checkCudaErrors(hipMemcpy(ns, nslots[r&1], NBUCKETS * sizeof(u32), hipMemcpyDeviceToHost));
    u32 binsizes[65];
    memset(binsizes, 0, 65 * sizeof(u32));
    for (u32 bucketid = 0; bucketid < NBUCKETS; bucketid++) {
      u32 bsize = min(ns[bucketid], NSLOTS) >> (SLOTBITS-6);
      binsizes[bsize]++;
    }
    for (u32 i=0; i < 65; i++) {
#ifdef HIST
      printf(" %d:%d", i, binsizes[i]);
#else
#ifdef SPARK
      u32 sparks = binsizes[i] / SPARKSCALE;
#else
      u32 sparks = 0;
      for (u32 bs = binsizes[i]; bs; bs >>= 1) sparks++;
      sparks = sparks * 7 / SPARKSCALE;
#endif
      printf("\342\226%c", '\201' + sparks);
#endif
    }
    printf("\n");
#endif
  }
  // proper dupe test is a little costly on GPU, so allow false negatives
  __device__ bool probdupe(u32 *prf) {
    unsigned short susp[PROOFSIZE];
    memset(susp, 0xffff, PROOFSIZE * sizeof(unsigned short));
    for (u32 i=0; i<PROOFSIZE; i++) {
      u32 bin = prf[i] & (PROOFSIZE-1);
      unsigned short msb = prf[i]>>WK;
      if (msb == susp[bin])
        return true;
      susp[bin] = msb;
    }
    return false;
  }
  struct htlayout {
    htalloc hta;
    u32 prevhashunits;
    u32 nexthashunits;
    u32 dunits;
    u32 prevbo;
    u32 nextbo;

    __device__ htlayout(equi *eq, u32 r): hta(eq->hta), prevhashunits(0), dunits(0) {
      u32 nexthashbytes = hashsize(r);
      nexthashunits = hashwords(nexthashbytes);
      prevbo = 0;
      nextbo = nexthashunits * sizeof(hashunit) - nexthashbytes; // 0-3
      if (r) {
        u32 prevhashbytes = hashsize(r-1);
        prevhashunits = hashwords(prevhashbytes);
        prevbo = prevhashunits * sizeof(hashunit) - prevhashbytes; // 0-3
        dunits = prevhashunits - nexthashunits;
      }
    }
    __device__ u32 getxhash0(const slot0* pslot) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] >> 4;
#elif WN == 200 && RESTBITS == 8
      return (pslot->hash->bytes[prevbo] & 0xf) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif WN == 144 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
      return (pslot->hash->bytes[prevbo] & 0x3) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#else
#error non implemented
#endif
    }
    __device__ u32 getxhash1(const slot1* pslot) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 8
      return pslot->hash->bytes[prevbo];
#elif WN == 144 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
      return pslot->hash->bytes[prevbo] &0x3f;
#else
#error non implemented
#endif
    }
    __device__ bool equal(const hashunit *hash0, const hashunit *hash1) const {
      return hash0[prevhashunits-1].word == hash1[prevhashunits-1].word;
    }
  };

  struct collisiondata {
#ifdef XBITMAP
#if NSLOTS > 64
#error cant use XBITMAP with more than 64 slots
#endif
    u64 xhashmap[NRESTS];
    u64 xmap;
#else
#if RESTBITS <= 6
    typedef uchar xslot;
#else
    typedef u16 xslot;
#endif
    static const xslot xnil = ~0;
    xslot xhashslots[NRESTS];
    xslot nextxhashslot[NSLOTS];
    xslot nextslot;
#endif
    u32 s0;

    __device__ void clear() {
#ifdef XBITMAP
      memset(xhashmap, 0, NRESTS * sizeof(u64));
#else
      memset(xhashslots, xnil, NRESTS * sizeof(xslot));
      memset(nextxhashslot, xnil, NSLOTS * sizeof(xslot));
#endif
    }
    __device__ bool addslot(u32 s1, u32 xh) {
#ifdef XBITMAP
      xmap = xhashmap[xh];
      xhashmap[xh] |= (u64)1 << s1;
      s0 = ~0;
      return true;
#else
      nextslot = xhashslots[xh];
      nextxhashslot[s1] = nextslot;
      xhashslots[xh] = s1;
      return true;
#endif
    }
    __device__ bool nextcollision() const {
#ifdef XBITMAP
      return xmap != 0;
#else
      return nextslot != xnil;
#endif
    }
    __device__ u32 slot() {
#ifdef XBITMAP
      const u32 ffs = __ffsll(xmap);
      s0 += ffs; xmap >>= ffs;
#else
      nextslot = nextxhashslot[s0 = nextslot];
#endif
      return s0;
    }
  };
};

__global__ void digitH(equi *eq) {
  uchar hash[HASHOUT];
  blake2b_state state;
  equi::htlayout htl(eq, 0);
  const u32 hashbytes = hashsize(0);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 block = id; block < NBLOCKS; block += eq->nthreads) {
    state = eq->blake_ctx;
    blake2b_gpu_hash(&state, block, hash, HASHOUT);
    for (u32 i = 0; i<HASHESPERBLAKE; i++) {
      const uchar *ph = hash + i * WN/8;
#if BUCKBITS == 16 && RESTBITS == 4
      const u32 bucketid = ((u32)ph[0] << 8) | ph[1];
#ifdef XINTREE
      const u32 xhash = ph[2] >> 4;
#endif
#elif BUCKBITS == 14 && RESTBITS == 6
      const u32 bucketid = ((u32)ph[0] << 6) | ph[1] >> 2;
#elif BUCKBITS == 12 && RESTBITS == 8
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#elif BUCKBITS == 20 && RESTBITS == 4
      const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 4) | ph[2] >> 4;
#ifdef XINTREE
      const u32 xhash = ph[2] & 0xf;
#endif
#elif BUCKBITS == 12 && RESTBITS == 4
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
      const u32 xhash = ph[1] & 0xf;
#else
#error not implemented
#endif
      const u32 slot = atomicAdd(&eq->nslots[0][bucketid], 1);
      if (slot >= NSLOTS)
        continue;
      slot0 &s = eq->hta.trees0[0][bucketid][slot];
#ifdef XINTREE
      s.attr = tree(block*HASHESPERBLAKE+i, xhash);
#else
      s.attr = tree(block*HASHESPERBLAKE+i);
#endif
      memcpy(s.hash->bytes+htl.nextbo, ph+WN/8-hashbytes, hashbytes);
    }
  }
}

__global__ void digitO(equi *eq, const u32 r) {
  equi::htlayout htl(eq, r);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        u32 xorbucketid;
        u32 xhash;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
        xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0xf) << 8)
                          | (bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1])) << 4
                  | (xhash = bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
        xhash &= 0xf;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 8)
                            | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 4)
                    | (xhash = bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 4;
        xhash &= 0xf;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 4)
                  | (xhash = bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
        xhash &= 0xf;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) & 0xf) << 8)
                           | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 2
                           | (bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 6;
#else
#error not implemented
#endif
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];
#ifdef XINTREE
        xs.attr = tree(bucketid, s0, s1, xhash);
#else
        xs.attr = tree(bucketid, s0, s1);
#endif
        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

__global__ void digitE(equi *eq, const u32 r) {
  equi::htlayout htl(eq, r);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
        xorbucketid = ((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) << 8)
                        | (bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]);
        u32 xhash = (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 8)
                            | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 4)
                            | (bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 4;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 4)
                          | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 6)
                          | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 2;
#else
#error not implemented
#endif
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];
#ifdef XINTREE
        xs.attr = tree(bucketid, s0, s1, xhash);
#else
        xs.attr = tree(bucketid, s0, s1);
#endif
        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

#ifdef UNROLL
// bucket mask
static const u32 BUCKMASK = NBUCKETS-1;

__global__ void digit_1(equi *eq) {
  equi::htlayout htl(eq, 1);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[0][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x0123);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[0][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
        xs.hash[4].word = pslot0->hash[5].word ^ pslot1->hash[5].word;
      }
    }
  }
}
__global__ void digit2(equi *eq) {
  equi::htlayout htl(eq, 2);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[0][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x0123);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[1][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[4].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
      }
    }
  }
}
__global__ void digit3(equi *eq) {
  equi::htlayout htl(eq, 3);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[1][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x1234);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[1][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
      }
    }
  }
}
__global__ void digit4(equi *eq) {
  equi::htlayout htl(eq, 4);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[1][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x4123);
        const u32 xorbucketid = bexor >> 8;
        const u32 xhash = bexor >> 4 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[2][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
      }
    }
  }
}
__global__ void digit5(equi *eq) {
  equi::htlayout htl(eq, 5);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[2][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
      }
    }
  }
}
__global__ void digit6(equi *eq) {
  equi::htlayout htl(eq, 6);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[3][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
      }
    }
  }
}
__global__ void digit7(equi *eq) {
  equi::htlayout htl(eq, 7);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[3][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x4012);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[3][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
      }
    }
  }
}
__global__ void digit8(equi *eq) {
  equi::htlayout htl(eq, 8);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[3][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x3456);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[4][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
      }
    }
  }
}
#endif

__global__ void digitK(equi *eq) {
  equi::collisiondata cd;
  equi::htlayout htl(eq, WK);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(WK-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid); // assume WK odd
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1))) // assume WK odd
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash)) {
#ifdef XINTREE
          eq->candidate(tree(bucketid, s0, s1, 0));
#else
          eq->candidate(tree(bucketid, s0, s1));
#endif
        }
      }
    }
  }
}

void runEquihash(int nthreads, int tpb, equi* eq, equi* device_eq, proof* sols) {

	printf("Digit 0\n");

	digitH<<<nthreads / tpb, tpb>>>(device_eq);

#if BUCKBITS == 16 && RESTBITS == 4 && defined XINTREE && defined(UNROLL)
	printf("Digit %d\n", 1);
	digit_1<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 2);
	digit2<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 3);
	digit3<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 4);
	digit4<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 5);
	digit5<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 6);
	digit6<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 7);
	digit7<<<nthreads / tpb, tpb>>>(device_eq);
	printf("Digit %d\n", 8);
	digit8<<<nthreads / tpb, tpb>>>(device_eq);
#else
	for (u32 r = 1; r < WK; r++) {
		printf("Digit %d\n", r);
		r & 1 ? digitO<<<nthreads / tpb, tpb>>>(device_eq, r)
			: digitE<<<nthreads / tpb, tpb>>>(device_eq, r);
		eq.showbsizes(r);
	}
#endif
	printf("Digit %d\n", WK);
	digitK<<<nthreads / tpb, tpb>>>(device_eq);
		
	checkCudaErrors(hipMemcpy(eq, device_eq, sizeof(equi), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(sols, eq->sols, MAXSOLS * sizeof(proof), hipMemcpyDeviceToHost));
}